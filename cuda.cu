#include <stdio.h>
#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>

// Compile with
// nvcc -O2 -std=c++11 cuda.cu

__global__ void
empty()
{

}


int
main(int argc, char **argv)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int threadsPerBlock = 256;
    int blocksPerGrid = 16;

    //Warmup
    empty<<<blocksPerGrid, threadsPerBlock>>>();
    empty<<<blocksPerGrid, threadsPerBlock>>>();
    empty<<<blocksPerGrid, threadsPerBlock>>>();
    
    int nRuns = 1000;

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    for(int i=0; i<nRuns; i++) {
        empty<<<blocksPerGrid, threadsPerBlock>>>();
    }

    std::chrono::steady_clock::time_point end= std::chrono::steady_clock::now();
        
    std::cout << (float)std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()/nRuns << " microseconds per call" <<std::endl;

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch empty kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

